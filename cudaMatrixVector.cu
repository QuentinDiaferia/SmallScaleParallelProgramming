#include "hip/hip_runtime.h"
#include <iostream>
#include "CSRMatrix.h"
#include "ELLPACKMatrix.h"
#include "mmio.h"
#include <ctime>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_timer.h>

using namespace std;

__global__ 
void CSRMult(const int *irp, const int* ja, const double* as, const double *v, double *res, const int rows) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = 0; i < rows; i++) {
		res[i] = 0;
		for (int j = irp[i]; j < irp[i + 1] - 1; j++) {
			res[i] += as[j] * v[ja[j]];
		}
	}
}

int main() {
	double time_ini, time_end, time_cpu;
	char* file = "matrices/cage4.mtx";

	// CONVERSION

	time_ini = clock();
	CSRMatrix m(file);
	time_end = clock();
	time_cpu = (time_end - time_ini) / CLOCKS_PER_SEC;
	cout << "Conversion to CSR : " << time_cpu << " seconds" << endl << endl;

	time_ini = clock();
	ELLPACKMatrix m2(file);
	time_end = clock();
	time_cpu = (time_end - time_ini) / CLOCKS_PER_SEC;
	cout << "Conversion to ELLPACK : " << time_cpu << endl;

	// CUDA

	// -- CSR

	int *irp, *ja;
	int *_irp, *_ja;
	double *as, *v, *result;
	double *_as, *_v, *_result;
	vector<int> vIrp = m.getIrp();
	vector<int> vJa = m.getJa();
	vector<double> vAs = m.getAs();

	irp = (int *)malloc(m.getRows() + 1 * sizeof(int));
	ja = (int *)malloc(m.getNz() * sizeof(int));
	as = (double *)malloc(m.getNz() * sizeof(double));
	v = (double *)malloc(m.getRows() * sizeof(double));
	result = (double *)malloc(m.getRows() * sizeof(double));

	for (int i = 0; i < m.getRows() + 1; i++) {
		irp[i] = vIrp[i];
		v[i] = 2;
		result[i] = 0;
	}
	for (int i = 0; i < m.getNz(); i++) {
		ja[i] = vJa[i];
		as[i] = vAs[i];
	}

	hipMalloc((void**)&_irp, sizeof(int) * m.getRows());
	hipMalloc((void**)&_ja, sizeof(int) * m.getNz());
	hipMalloc((void**)&_as, sizeof(double) * m.getNz());
	hipMalloc((void**)&_v, sizeof(double) * m.getRows());
	hipMalloc((void**)&_result, sizeof(double) * m.getRows());

	hipMemcpy(_irp, irp, sizeof(int) * m.getRows(), hipMemcpyHostToDevice);
	hipMemcpy(_ja, ja, sizeof(int) * m.getNz(), hipMemcpyHostToDevice);
	hipMemcpy(_as, as, sizeof(double) * m.getNz(), hipMemcpyHostToDevice);
	hipMemcpy(_v, v, sizeof(double) * m.getRows(), hipMemcpyHostToDevice);
	hipMemcpy(_result, result, sizeof(double) * m.getRows(), hipMemcpyHostToDevice);

	int BLOCK_DIM = 256;

	/*StopWatchInterface* timer = 0;
	sdkCreateTimer(&timer);
	timer->start();*/
	CSRMult << <m.getNz(), BLOCK_DIM >> >(_irp, _ja, as, _v, _result, m.getRows());

	hipDeviceSynchronize();
	//timer->stop();

	hipMemcpy(result, _result, sizeof(double) * m.getRows(), hipMemcpyDeviceToHost);

	printf("\n");
	//cout << "timer: " << timer->getTime() << std::endl;
	printf("\n");

	hipFree(_irp);
	hipFree(_ja);
	hipFree(_as);
	hipFree(_v);
	hipFree(_result);

	free(irp);
	free(ja);
	free(as);
	free(v);
	free(result);

	return 0;
}