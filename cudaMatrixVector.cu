#include "hip/hip_runtime.h"
#include <iostream>
#include "CSRMatrix.h"
#include "ELLPACKMatrix.h"
#include "mmio.h"
#include <ctime>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>

using namespace std;

__global__ 
void CSRMult(const int *irp, const int* ja, const double* as, const double *v, double *result, const int rows) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < rows) {
		double sum = 0;
		for (int j = irp[i]; j < irp[i + 1]; j++) {
			sum += as[j] * v[ja[j-1]];
		}
		result[i] = sum ;
	}
}

__global__ 
void ELLPACKMult(const int maxnz, const int* ja, const double* as, const double *v, double *res, const int rows) {
	/*
	for (int i = 0; i < rows; i++) {
		res[i] = 0;
		for (int j = 0; j < maxnz; j++) {
			res[i] += as[i][j] * v[ja[i][j]];
		}
	}
	*/
}

int main() {
	double time_ini, time_end, time_cpu, total_time;
	char* file = "matrices/cage4.mtx";

	// CONVERSION

	time_ini = clock();
	CSRMatrix m(file);
	time_end = clock();
	time_cpu = (time_end - time_ini) / CLOCKS_PER_SEC;
	cout << "Conversion to CSR : " << time_cpu << " seconds" << endl << endl;

	time_ini = clock();
	ELLPACKMatrix m2(file);
	time_end = clock();
	time_cpu = (time_end - time_ini) / CLOCKS_PER_SEC;
	cout << "Conversion to ELLPACK : " << time_cpu << endl;

	// CUDA

	int *irp, *ja;
	int *_irp, *_ja;
	double *as, *v, *result;
	double *_as, *_v, *_result;
	vector<int> vIrp = m.getIrp();
	vector<int> vJa = m.getJa();
	vector<double> vAs = m.getAs();


	irp = (int *)malloc((m.getRows() + 1 ) * sizeof(int));
	ja = (int *)malloc(m.getNz() * sizeof(int));
	as = (double *)malloc(m.getNz() * sizeof(double));
	v = (double *)malloc(m.getRows() * sizeof(double));
	result = (double *)malloc(m.getRows() * sizeof(double));

	// Vector<> to simple arrays

	for (int i = 0; i < m.getRows() + 1; i++) {
		irp[i] = vIrp[i];
		if (i < m.getRows()) {
			v[i] = 2.0;
			result[i] = 0.0;
		}
	}
	for (int i = 0; i < m.getNz(); i++) {
		ja[i] = vJa[i];
		as[i] = vAs[i];
	}

	hipMalloc((void**)&_irp, sizeof(int) * (m.getRows() + 1));
	hipMalloc((void**)&_ja, sizeof(int) * m.getNz());
	hipMalloc((void**)&_as, sizeof(double) * m.getNz());
	hipMalloc((void**)&_v, sizeof(double) * m.getRows());
	hipMalloc((void**)&_result, sizeof(double) * m.getRows());

	hipMemcpy(_irp, irp, sizeof(int) * (m.getRows() + 1), hipMemcpyHostToDevice);
	hipMemcpy(_ja, ja, sizeof(int) * m.getNz(), hipMemcpyHostToDevice);
	hipMemcpy(_as, as, sizeof(double) * m.getNz(), hipMemcpyHostToDevice);
	hipMemcpy(_v, v, sizeof(double) * m.getRows(), hipMemcpyHostToDevice);
	hipMemcpy(_result, result, sizeof(double) * m.getRows(), hipMemcpyHostToDevice);

	int BLOCK_DIM = 128;
	int GRID_DIM = m.getRows() / 128 + 1;

	total_time = 0.0;
	for (int i = 0; i < 10; i++) {

		time_ini = clock();

		CSRMult<<<GRID_DIM, BLOCK_DIM>>>(_irp, _ja, as, _v, _result, m.getRows());
		hipDeviceSynchronize();

		time_end = clock();
		time_cpu = (time_end - time_ini) / CLOCKS_PER_SEC;
		total_time += time_cpu;
	}

	total_time /= 10;

	hipMemcpy(result, _result, sizeof(double) * m.getRows(), hipMemcpyDeviceToHost);

	for (int i = 0; i < m.getRows(); i++)
		cout << result[i] << endl;

	cout << endl << "average time  : " << total_time << endl;
	cout << "FLOPS  : " << 2 * m.getNz() / total_time << endl << endl;



	hipFree(_irp);
	hipFree(_ja);
	hipFree(_as);
	hipFree(_v);
	hipFree(_result);

	free(irp);
	free(ja);
	free(as);
	free(v);
	free(result);

	return 0;
}